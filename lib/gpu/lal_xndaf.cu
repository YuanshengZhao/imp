#include "hip/hip_runtime.h"
// **************************************************************************
//                                   xndaf.cu
//                             -------------------
//                           Trung Dac Nguyen (ORNL)
//
//  Device code for acceleration of the xndaf pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : nguyentd@ornl.gov
// ***************************************************************************

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
#else
_texture_2d( pos_tex,int4);
#endif
#else
#define pos_tex x_
#endif

__kernel void k_xndaf(const __global numtyp4 *restrict x_,
                    const numtyp cutsq,
                    const numtyp drinv,
                    const int n_table,
                    const __global int *restrict tabindex,
                    const __global numtyp *restrict frc_tb,
                    const int lj_types,
                    const __global numtyp *restrict sp_lj_in,
                    const __global int *dev_nbor,
                    const __global int *dev_packed,
                    __global acctyp4 *restrict ans,
                    __global acctyp *restrict engv,
                    const int eflag, const int vflag, const int inum,
                    const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_lj[4];
  int n_stride;
  local_allocate_store_pair();

  sp_lj[0]=sp_lj_in[0];
  sp_lj[1]=sp_lj_in[1];
  sp_lj[2]=sp_lj_in[2];
  sp_lj[3]=sp_lj_in[3];

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<cutsq) {
        rsq=ucl_sqrt(rsq);
        int tbi=(int)(rsq*drinv);
        numtyp force = (numtyp)0;
        if(tbi<n_table) {
          force=frc_tb[tbi+tabindex[mtype]]*factor_lj;
          f.x+=delx*force;
          f.y+=dely*force;
          f.z+=delz*force;
        }
      }

    } // for nbor
  } // if ii
  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
}

__kernel void k_xndaf_fast(const __global numtyp4 *restrict x_,
                         const numtyp cutsq,
                         const numtyp drinv,
                         const int n_table,
                         const __global int *restrict tabindex,
                         const __global numtyp *restrict frc_tb,
                         const __global numtyp *restrict sp_lj_in,
                         const __global int *dev_nbor,
                         const __global int *dev_packed,
                         __global acctyp4 *restrict ans,
                         __global acctyp *restrict engv,
                         const int eflag, const int vflag, const int inum,
                         const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_lj[4];
  int n_stride;
  local_allocate_store_pair();

  if (tid<4)
    sp_lj[tid]=sp_lj_in[tid];

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutsq) {
        rsq=ucl_sqrt(rsq);
        int tbi=(int)(rsq*drinv);
        numtyp force = (numtyp)0;
        if(tbi<n_table) {
          force=frc_tb[tbi+tabindex[mtype]]*factor_lj;
          f.x+=delx*force;
          f.y+=dely*force;
          f.z+=delz*force;
        }
      }

    } // for nbor
  } // if ii
  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
}
